#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#include "common.cu"

#include "shaders/lambert.cu"
#include "shaders/specular.cu"
#include "shaders/cook_torrance.cu"


// CONFIGURATION
#define STREAM_COMPACTION 1
#define MATERIAL_SORTING 0  // enable this if you have a high number of materials

// Bump the shader version to recompile shaders. We need a better solution for this
#define SHADER_VER 2.8

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
        segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.kill = false;

        
        CREATE_RANDOM_ENGINE(iter, index, traceDepth, u01, rng);

        float x1 = u01(rng) - 0.5f;
        float x2 = u01(rng) - 0.5f;

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float(x) + x1) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float(y) + x2) - (float)cam.resolution.y * 0.5f)
        );

        segment.ray.inv_direction = glm::vec3(1.0f) / segment.ray.direction;

        segment.ray.sign.x = (segment.ray.inv_direction.x < 0) ? 1 : 0;
        segment.ray.sign.y = (segment.ray.inv_direction.y < 0) ? 1 : 0;
        segment.ray.sign.z = (segment.ray.inv_direction.z < 0) ? 1 : 0;

        segment.pixelIndex = index;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == MESH)
            {
                t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__global__ void advancePathSegments(int num_paths, PathSegment* paths, ShadeableIntersection *intersections) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths)
    {
        return;
    }

    if (intersections[idx].t == -1.0f && !paths[idx].hitEmissive) {
        paths[idx].kill = true;
        return;
    }

    Ray &ray = paths[idx].ray;

    ray.origin = getPointOnRay(ray, intersections[idx].t);
    ray.direction = paths[idx].sample_dir;
    ray.inv_direction = glm::vec3(1.0f) / ray.direction;

    ray.sign.x = (ray.inv_direction.x < 0) ? 1 : 0;
    ray.sign.y = (ray.inv_direction.y < 0) ? 1 : 0;
    ray.sign.z = (ray.inv_direction.z < 0) ? 1 : 0;
}

__global__ void shadePath(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    int depth
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection &intersection = shadeableIntersections[idx];
        PathSegment &path = pathSegments[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material &material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            if (material.material_type == MaterialType::Emissive) {
                path.color += path.throughput * material.emittance * material.color;
                path.kill = true;
            } 
            else if (material.material_type == MaterialType::Diffuse) {
                Lambert::shadePathLambert(idx, iter, num_paths, depth, intersection, path, material);
            } 
            else if (material.material_type == MaterialType::Specular) {
                PerfectSpecular::shadePathSpecular(path, material);
            }
            else if (material.material_type == MaterialType::Microfacet) {
                CookTorrance::shadePathCookTorrance(intersection, path, material);
            }
        }
    }
}  


__global__ void getSampleDir(int num_paths, int iter, int depth, PathSegment* paths, ShadeableIntersection *intersections, Material *materials) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths)
    {
        return;
    }

    ShadeableIntersection &intersection = intersections[idx];
    PathSegment &path = paths[idx];

    if (intersection.t > 0.0f)
    {
        Material &material = materials[intersection.materialId];
        if (material.material_type == MaterialType::Emissive || material.material_type == MaterialType::Diffuse) {
            Lambert::sampleHemisphere(idx, num_paths, iter, depth, path, intersection);
        } 
        else if (material.material_type == MaterialType::Specular) {
            PerfectSpecular::sampleMirror(path, intersection);
        }
        else if (material.material_type == MaterialType::Microfacet) {
            CookTorrance::sampleCookTorrance(path, material, idx, iter, depth, -path.ray.direction, intersection.surfaceNormal, material.roughness);
        }
    }
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        glm::vec3 color = iterationPath.color;

        //reinhard tonemap
        color = color / (color + glm::vec3(1.0f));

        //gamma correction
        color = glm::pow(color, glm::vec3(1.0f / 2.2f));

        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}


// for stream compaction
struct path_terminated {
    __host__ __device__ bool operator()(PathSegment &path) const {
        return !path.kill;
    }
};

struct sort_materials {
    __host__ __device__ bool operator()(const ShadeableIntersection &sA, const ShadeableIntersection &sB) const {
        return sA.materialId < sB.materialId;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = BLOCK_SIZE_1D;

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    int num_paths = pixelcount;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // if (iter == 1) {
            // printf("NumPaths: %d\n", num_paths);
        // }

        // clean shading chunks
        hipMemset(dev_intersections, 0, num_paths * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("compute intersections");
        depth++;

        #if MATERIAL_SORTING
            thrust::sort_by_key(
                thrust::device,
                dev_intersections,
                dev_intersections + num_paths,
                dev_paths,
                sort_materials()
            );
        #endif

        getSampleDir<<<numblocksPathSegmentTracing, blockSize1d>>> (
            num_paths, 
            iter, 
            depth, 
            dev_paths, 
            dev_intersections,
            dev_materials
        );
        checkCUDAError("sample hemisphere");

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadePath<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            depth
        );

        if (depth == traceDepth) {
            iterationComplete = true; // TODO: should be based off stream compaction results.
        }

        advancePathSegments<<<numblocksPathSegmentTracing, blockSize1d>>>(
            num_paths,
            dev_paths,
            dev_intersections
        );
        checkCUDAError("advance path segments");

        #if STREAM_COMPACTION
            auto new_end = thrust::partition(dPtr(dev_paths), dPtr(dev_paths) + num_paths, path_terminated());
            num_paths = new_end - dPtr(dev_paths);
            checkCUDAError("thrust::remove_if");
        #endif // STREAM_COMPACTION

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }

        if (num_paths == 0) {
            iterationComplete = true;
        }
    }

        // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);


    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
