#include "mesh.h"

void Mesh::make_mesh_host(std::vector<glm::vec3> v, std::vector<int> indices) {
    num_verts = v.size();
    num_indices = indices.size();

    h_verts = v;
    h_indices = indices;
    h_valid = true;
}

void Mesh::make_mesh_device() {
    hipMalloc((void**)&d_verts, num_verts * sizeof(glm::vec3));
    hipMemcpy(d_verts, h_verts.data(), num_verts * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_indices, num_indices * sizeof(int));
    hipMemcpy(d_indices, h_indices.data(), num_indices * sizeof(int), hipMemcpyHostToDevice);
    d_valid = true;
}

void Mesh::delete_mesh_device() {
    hipFree(d_verts);
    hipFree(d_indices);
    d_valid = false;
}