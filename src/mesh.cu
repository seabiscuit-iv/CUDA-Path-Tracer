#include "mesh.h"

void Mesh::make_mesh_host(std::vector<glm::vec3> v) {
    num_verts = v.size();
    h_verts = v;
    // hipMalloc((void**)&d_verts, num_verts * sizeof(glm::vec3));
    // hipMemcpy(d_verts, v.data(), num_verts * sizeof(glm::vec3), hipMemcpyHostToDevice);
    h_valid = true;
}

void Mesh::make_mesh_device() {
    hipMalloc((void**)&d_verts, num_verts * sizeof(glm::vec3));
    hipMemcpy(d_verts, h_verts.data(), num_verts * sizeof(glm::vec3), hipMemcpyHostToDevice);
    d_valid = true;
}

void Mesh::delete_mesh_device() {
    hipFree(d_verts);
    d_valid = false;
}