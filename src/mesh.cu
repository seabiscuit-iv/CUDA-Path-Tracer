#include "hip/hip_runtime.h"
#include "mesh.h"
#include <stack>
#include "stack.h"
#include "sceneStructs.h"

void Mesh::make_mesh_host(std::vector<glm::vec3> v, std::vector<int> indices, std::vector<glm::vec3> normals, std::vector<int> normal_indices) {
    num_verts = v.size();
    num_indices = indices.size();
    num_normals = normals.size();
    num_normal_indices = normal_indices.size();

    h_verts = v;
    h_indices = indices;

    if (normals.size() > 0 && normal_indices.size() > 0) {
        h_normals = normals;
        h_normal_indices = normal_indices;
        has_normal_buffers = true;
    }

    h_valid = true;
}

void Mesh::make_mesh_device() {
    hipMalloc((void**)&d_verts, num_verts * sizeof(glm::vec3));
    hipMemcpy(d_verts, h_verts.data(), num_verts * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_indices, num_indices * sizeof(int));
    hipMemcpy(d_indices, h_indices.data(), num_indices * sizeof(int), hipMemcpyHostToDevice);
    
    if (has_normal_buffers) {
        hipMalloc((void**)&d_normals, num_normals * sizeof(glm::vec3));
        hipMemcpy(d_normals, h_normals.data(), num_normals * sizeof(glm::vec3), hipMemcpyHostToDevice);
        
        hipMalloc((void**)&d_normal_indices, num_normal_indices * sizeof(int));
        hipMemcpy(d_normal_indices, h_normal_indices.data(), num_normal_indices * sizeof(int), hipMemcpyHostToDevice);
    }

    bvh.make_bvh(h_verts, h_indices);

    d_valid = true;
}

void Mesh::delete_mesh_device() {
    hipFree(d_verts);
    hipFree(d_indices);
    
    if (has_normal_buffers) {
        hipFree(d_normals);
        hipFree(d_normal_indices);
    }

    bvh.delete_bvh();

    d_valid = false;
}


void BVH::delete_bvh() {
    hipFree(dev_bvh);
    initizalized = false;
}


// START AND END ARE INCLUSIVE
// we access indicies with 3 * start or 3 * end
BoundingBox fill_bvh(int idx, int start, int end, std::vector<BVHNode> &h_bvh, const std::vector<glm::vec3> &verts, const std::vector<int> &indices) {
    if (start == end) {
        // base case - single triangle
        h_bvh[idx] = BVHNode();
        h_bvh[idx].make_bvh_leaf_node(start);
        glm::vec3 local_tri[3] = {
            verts[indices[3 * start]],
            verts[indices[3 * start + 1]],
            verts[indices[3 * start + 2]]
        };
        return BoundingBox(local_tri, 3);
    }

    // partition [start_a ... end_a][start_b ... end_b]

    // currently we have 9
    // want to divide by 2 and allocate anything additionally after the fact
    // log2(9) - 1 = 2
    // partition = 4
    // 9 - 2*4 = 1
    // 4 + min(4, 1)
    // 4 + min (0, 1-4)

    // 14
    // log2(14) - 1 = 2
    // partition = 4
    // 14 - 2*4 = 6
    // 4 + min(4, 6) -> rem => 8
    // 4 + 6 - rem = 6

    int num_leaves = (end - start + 1);
    int log = glm::floor(glm::log2(float(num_leaves))) - 1.0f;
    int partition = glm::pow(2, log);
    int rem = num_leaves - 2 * partition;
    int left_alloc = min(rem, partition);
    int right_alloc = rem - left_alloc;

    int window_size_a = partition + left_alloc;
    int window_size_b = partition + right_alloc;

    int start_a = start;
    int end_a = start + window_size_a - 1;
    int start_b = end_a + 1;
    int end_b = end;

    if ( end_b - start_b + 1 != window_size_b ) {
        printf("Error: end_b - start_b (%d) != right_alloc (%d)\n", end_b - start_b + 1, right_alloc);
    }

    if ( window_size_a < window_size_b ) {
        printf("Window Size A %d was smaller than Window Size B %d\n", window_size_a, window_size_b);
    }

    BoundingBox bbox_a = fill_bvh(LEFT_NODE(idx), start_a, end_a, h_bvh, verts, indices);
    BoundingBox bbox_b = fill_bvh(RIGHT_NODE(idx), start_b, end_b, h_bvh, verts, indices);

    glm::vec3 combined_bbox[4] = {
        bbox_a.box_max,
        bbox_a.box_min,
        bbox_b.box_max,
        bbox_b.box_min,
    };

    h_bvh[idx] = BVHNode();
    auto fin_bbox = BoundingBox(combined_bbox, 4);
    h_bvh[idx].make_bvh_node(fin_bbox);

    return fin_bbox;
}



void BVH::make_bvh(std::vector<glm::vec3> verts, std::vector<int> indices) {
    // a binary tree with n leaf nodes must have 2n-1 nodes
    int num_leafs = indices.size() / 3; // 3 indices per triangle
    num_nodes = 2 * num_leafs - 1;

    std::vector<BVHNode> h_bvh(num_nodes);

    fill_bvh(0, 0, num_leafs-1, h_bvh, verts, indices);

    hipMalloc((void**)&dev_bvh, num_nodes * sizeof(BVHNode));
    hipMemcpy(dev_bvh, h_bvh.data(), num_nodes * sizeof(BVHNode), hipMemcpyHostToDevice);

    initizalized = true;
}

__host__ __device__
bool BoundingBox::RayBoxInterection(Ray ray) {
    float tmin = -FLT_MAX;
    float tmax =  FLT_MAX;

    ray.direction = glm::normalize(ray.direction);

    for (int i = 0; i < 3; i++) {
        if (fabs(ray.direction[i]) < 1e-8f) {
            if (ray.origin[i] < box_min[i] || ray.origin[i] > box_max[i]) {
                return false;
            }
        } 
        else {
            float invD = 1.0f / ray.direction[i];
            float t0 = (box_min[i] - ray.origin[i]) * invD;
            float t1 = (box_max[i] - ray.origin[i]) * invD;
            if (t0 > t1) {
                float temp = t0;
                t0 = t1;
                t1 = temp;
            }

            tmin = glm::max(tmin, t0);
            tmax = glm::min(tmax, t1);

            if (tmax < tmin) {
                return false;
            }
        }
    }

    // tmin_out = tmin;
    // tmax_out = tmax;
    return true;
}